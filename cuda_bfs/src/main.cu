#include <iostream>
#include <queue>
#include "kernels.cuh"
#include "bfs_prefix_scan.cuh"
#include "graph.hpp"
#include "hip/hip_runtime.h"
#include "scan.cuh"
#include ""
#include "bfs_layer_count.cuh"



void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev);
void cpu_BFS(const Graph& g, int start, int end);
int main(int argc, char** argv) {
    const char *path = "data/wiki-topcats.txt";
    int start = 120;
    int end = 1132332;
    if(argc == 4 || argc == 5) {
        path = argv[1];
        start = atoi(argv[2]);
        end = atoi(argv[3]);
    }
    Graph new_graph = get_Graph_from_file(path);
    cpu_BFS(new_graph,start,end);

    hipSetDevice(0);
    cuda_BFS_prefix_scan(new_graph, start, end);
    cuda_BFS_frontier_numbers(new_graph,start,end);

    //check output
    if(argc == 5) check_output(new_graph,start,end);

    return 0;
}

void compute_bfs(const Graph& g, int start, int end, std::vector<int>& prev) {
    std::vector<bool> visited(g.n);
    std::queue<int> Q;

    Q.push(start);
    visited[start] = true;

    //start measure time
    hipEvent_t start_time,stop_time;
    float time;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);
    hipEventRecord(start_time,0);

    while(!Q.empty()) {
        int v = Q.front();
        Q.pop();

        if(visited[end]) break;

        int neighbours_count = g.v_adj_length[v];
        int neighbours_offset = g.v_adj_begin[v];
        for(int i =0; i<neighbours_count; i++) {
            int neighbour = g.v_adj_list[neighbours_offset + i];

            if(!visited[neighbour]) {
                visited[neighbour] = true;
                prev[neighbour] = v;
                Q.push(neighbour);

                if(neighbour == end) {
                    break;
                }
            }
        }
    }
    //end measure time
    hipEventRecord(stop_time,0);
    hipEventSynchronize(stop_time);
    hipEventElapsedTime(&time,start_time,stop_time);
    hipEventDestroy(start_time);
    hipEventDestroy(stop_time);
    std::cout<<"cpu bfs took: "<<time <<" ms\n";

}

void cpu_BFS(const Graph &g, int start, int end) {
    std::vector<int> prev(g.n);
    for(int v = 0; v<g.n; v++) {
        prev[v] = UINT_MAX;
    }
    compute_bfs(g,start,end,prev);

    get_path(start,end,prev.data(),g.n,"output/cpu_output.txt");
}