#include "hip/hip_runtime.h"
#include "bfs_layer_count.cuh"
__global__ void kernel_cuda_frontier_numbers(int *v_adj_list, int *v_adj_begin, int *v_adj_length,
        int n, int *result, int* prev, bool *still_running, int end, int iteration) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    //if I would call not enough threads
    for (int v = 0; v < n; v += num_threads)
    {
        int vertex = v + tid;
        if (vertex < n && result[vertex] == iteration)
        {
            for (int i = 0; i < v_adj_length[vertex]; i++)
            {
                int neighbor = v_adj_list[v_adj_begin[vertex] + i];

                //check if not visited yet
                if (result[neighbor] == n + 1)
                {
                    result[neighbor] = iteration + 1;
                    prev[neighbor] = vertex;

                    if(neighbor == end) {

                        *still_running = false;
                        break;
                    }
                    *still_running = true; // we added neighbour to queue
                }

            }
        }
    }
}

void cuda_BFS_frontier_numbers(const Graph& G, int start, int end) {
    int* v_adj_list;
    int* v_adj_begin;
    int* v_adj_length;
    int* result;
    int* prev;

    int* h_result = (int*)malloc(G.n * sizeof(int));

    bool* running;
    int level = 0;

    hipMalloc(&v_adj_list, sizeof(int) * G.m);
    hipMalloc(&v_adj_begin, sizeof(int) * G.n);
    hipMalloc(&v_adj_length, sizeof(int) * G.n);
    hipMalloc(&prev, sizeof(int) * G.n);
    hipMalloc(&result,sizeof(int) * G.n);
    hipMalloc(&running, sizeof(bool) * 1);

    const int THREADS_PER_BLOCK = 512;
    int blocks = G.n / THREADS_PER_BLOCK;
    if(blocks == 0) blocks = 1;


    std::fill_n(h_result,G.n,G.n + 1);
    h_result[start] = 0;

    hipMemcpy(v_adj_list, G.v_adj_list.data(), sizeof(int) * G.m, hipMemcpyHostToDevice);
    hipMemcpy(v_adj_begin, G.v_adj_begin.data(), sizeof(int) * G.n, hipMemcpyHostToDevice);
    hipMemcpy(v_adj_length, G.v_adj_length.data(), sizeof(int) * G.n, hipMemcpyHostToDevice);
    hipMemcpy(result, h_result, sizeof(int) * G.n, hipMemcpyHostToDevice);
    hipMemcpy(prev,h_result,sizeof(int) * G.n,hipMemcpyHostToDevice);
    bool* h_running = new bool[1];

    //start measuring time
    hipEvent_t start_time,stop_time;
    float time;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);
    hipEventRecord(start_time,0);

    do
    {
        *h_running = false;
        hipMemcpy(running, h_running, sizeof(bool) * 1, hipMemcpyHostToDevice);

        kernel_cuda_frontier_numbers<<<blocks, THREADS_PER_BLOCK>>>(v_adj_list,v_adj_begin,v_adj_length,
                                                      G.n,result,prev,running,
                                                      end,level);

        level++;
        hipMemcpy(h_running, running, sizeof(bool) * 1, hipMemcpyDeviceToHost);
    } while (*h_running);

    hipEventRecord(stop_time,0);
    hipEventSynchronize(stop_time);
    hipEventElapsedTime(&time,start_time,stop_time);
    hipEventDestroy(start_time);
    hipEventDestroy(stop_time);
    std::cout<<"gpu bfs with layer counter took: "<<time <<" ms\n";

    //copy prev array to cpu
    int* h_prev = (int*)malloc(G.n * sizeof(int));
    hipMemcpy(h_prev,prev,G.n * sizeof(int),hipMemcpyDeviceToHost);
    get_path(start,end,h_prev,G.n,"output/gpu_layer_output.txt");

    hipFree(v_adj_list);
    hipFree(v_adj_begin);
    hipFree(v_adj_length);
    hipFree(prev);
    hipFree(result);
    hipFree(running);


    free(h_prev);
    free(h_result);
    free(h_running);
}