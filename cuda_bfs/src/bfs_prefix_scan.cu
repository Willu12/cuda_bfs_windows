#include "hip/hip_runtime.h"
#include "bfs_prefix_scan.cuh"

hipError_t cuda_init(const Graph& G, int** v_adj_list, int** v_adj_begin, int** v_adj_length,int** queue,
                      int** prev,bool** visited, int** frontier,int** prefix_scan) {

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)v_adj_list, G.m * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)v_adj_begin, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)v_adj_length, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(queue, (G.n + 1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)prev, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc((void**)frontier, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = cuda_calloc(visited, G.n * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = cuda_calloc(prefix_scan, G.n * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_list, G.v_adj_list.data(), G.m * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*(void**)v_adj_begin, G.v_adj_begin.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(*(void**)v_adj_length, G.v_adj_length.data(), G.n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    // cuda_free_all(*v_adj_list,*v_adj_begin, *v_adj_length, *queue, *prev, *visited, *frontier, *prefix_scan);

    return cudaStatus;
}

inline hipError_t cuda_calloc( void *devPtr, size_t size ) {
    hipError_t err = hipMalloc( (void**)devPtr, size );
    if( err == hipSuccess ) err = hipMemset( *(void**)devPtr, 0, size );
    return err;
}

hipError_t cuda_prefix_scan(int* frontier, int** prefix_scan, int n) {
    hipError_t err = hipMemset( *(void**)prefix_scan, 0, n * sizeof(int) );
    if(err != hipSuccess) return err;
    scan(*prefix_scan,frontier,n);
    return err;
}

void queue_from_prefix(int* prefix_scan, int* queue,int* frontier, int n) {
    const int THREADS_PER_BLOCK = 512;
    int blocks = n / THREADS_PER_BLOCK;
    if(blocks == 0) blocks = 1;
    queue_from_prescan<<<blocks,THREADS_PER_BLOCK>>>(queue, prefix_scan, frontier,n);
}


hipError_t create_queue(int* frontier,int** prefix_scan, int** queue,int n) {
    //clear previous queue
    hipError_t err;

    if(hipSuccess != (err = hipMemset( *(void**)queue, 0, n * sizeof(int)) )) return err;

    if(hipSuccess != (err = cuda_prefix_scan(frontier,prefix_scan,n))) return err;

    queue_from_prefix(*prefix_scan,*queue,frontier,n);
    return err;
}

void cuda_prefix_queue_iter(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,bool* visited,int*frontier,int* prev,int end,
                            bool* d_stop,bool* h_stop,int n) {
    const int THREADS_PER_BLOCK = 512;
    int queue_length = 0;

    hipMemcpy(&queue_length,queue,sizeof(int),hipMemcpyDeviceToHost);
    if(queue_length == 0) {
        *h_stop = true;
        return;
    }
    //amount of blocks with ceil
    int blocks = queue_length / THREADS_PER_BLOCK + !!(queue_length % THREADS_PER_BLOCK);
    bfs_cuda_prescan_iter<<<blocks,THREADS_PER_BLOCK>>>(v_adj_list,v_adj_begin,v_adj_length,queue,frontier,visited,prev,end,d_stop,n);
    hipMemcpy(h_stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost);
}

void cuda_free_all(int* v_adj_list, int* v_adj_begin, int* v_adj_length,int* queue,
                   int* prev,bool* visited, int* frontier,int* prefix_scan) {
    hipFree(v_adj_list);
    hipFree(v_adj_begin);
    hipFree(v_adj_length);
    hipFree(queue);
    hipFree(prev);
    hipFree(visited);
    hipFree(frontier);
    hipFree(prefix_scan);
}

hipError_t cuda_BFS_prefix_scan(const Graph& G, int start, int end) {
    int* v_adj_list = nullptr;
    int* v_adj_begin = nullptr;
    int* v_adj_length = nullptr;
    int* queue = nullptr;
    int* prev = nullptr;
    int* prefix_scan = nullptr;
    bool* visited = nullptr;
    int* frontier = nullptr;
    hipError_t cudaStatus;

    bool stop = false;
    bool* d_stop;
    hipMalloc(&d_stop,sizeof(bool));
    hipMemset(d_stop,0,sizeof(bool));

    cudaStatus = cuda_init(G,&v_adj_list,&v_adj_begin,&v_adj_length,&queue,&prev,&visited,&frontier,&prefix_scan);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda init failed");
    }
    int* host_queue = (int*)malloc(sizeof(int) * 2);
    host_queue[0] = 1;
    host_queue[1] = start;

    hipMemcpy(queue,host_queue,2 * sizeof(int),hipMemcpyHostToDevice);
    free(host_queue);


    hipEvent_t start_time,stop_time;
    float time;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);
    hipEventRecord(start_time,0);
    //main loop
    while(!stop) {

        //iter
        cuda_prefix_queue_iter(v_adj_list,v_adj_begin,v_adj_length,queue,visited,frontier,prev,end,d_stop,&stop,G.n);
        //create queue
        create_queue(frontier,&prefix_scan,&queue,G.n);
        //clear frontier
        cudaStatus = hipMemset(frontier, 0, G.n * sizeof(int));
        //bfs layer scan
    }

    hipEventRecord(stop_time,0);
    hipEventSynchronize(stop_time);
    hipEventElapsedTime(&time,start_time,stop_time);
    hipEventDestroy(start_time);
    hipEventDestroy(stop_time);
    std::cout<<"gpu bfs with prefix_scan took: "<<time <<" ms\n";


    //copy prev array to cpu
    int* h_prev = (int*)malloc(G.n * sizeof(int));
    hipMemcpy(h_prev,prev,G.n * sizeof(int),hipMemcpyDeviceToHost);
    cuda_free_all(v_adj_list,v_adj_begin, v_adj_length, queue, prev, visited, frontier, prefix_scan);
    hipFree(d_stop);

    get_path(start,end,h_prev,G.n,"output/gpu_output.txt");
    free(h_prev);
    return cudaStatus;
}