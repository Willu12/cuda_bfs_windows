#include "hip/hip_runtime.h"
#include "kernels.cuh"
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo)
{
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	
	if (threadID < n) {
		temp[ai + bankOffsetA] = input[ai];
		temp[bi + bankOffsetB] = input[bi];
	}
	else {
		temp[ai + bankOffsetA] = 0;
		temp[bi + bankOffsetB] = 0;
	}
	

	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) {
		temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // clear the last element
	}

	for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[ai] = temp[ai + bankOffsetA];
		output[bi] = temp[bi + bankOffsetB];
	}
}

__global__ void prescan_large(int *output, int *input, int n, int *sums) {
	extern __shared__ int temp[];

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;
	
	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = input[blockOffset + ai];
	temp[bi + bankOffsetB] = input[blockOffset + bi];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) { 
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	} 
	
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + ai] = temp[ai + bankOffsetA];
	output[blockOffset + bi] = temp[bi + bankOffsetB];
}


__global__ void add(int *output, int length, int *n) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

__global__ void queue_from_prescan(int* queue,int* prefix_sum,int* frontier,int n) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(int v = 0; v < n; v+=num_threads){
        int vertex = v + tid;
        if (vertex < n && frontier[vertex]) queue[prefix_sum[vertex] + 1] = vertex;
    }

 	//size of queue
 	if (tid == 0) queue[0] = prefix_sum[n -1] + (int) frontier[n -1];
}

__global__ void bfs_cuda_prescan_iter(int* v_adj_list,int* v_adj_begin,int* v_adj_length,int* queue, int* frontier, bool* visited,
									  int *prev ,int end, bool* stop,int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= n) return;
	int v = queue[tid + 1];
	int offset = v_adj_begin[v];
	for(int i =0; i<v_adj_length[v]; i++) {
       // if(visited[end]) break;
		int u = v_adj_list[offset + i];
        if(visited[u]) continue;
        frontier[u] = 1;
		visited[u] = true;
		prev[u] = v;
		if(u == end){
			*stop = true;
			break;
		}
	}
}

__global__ void init_frontier(int* frontier, int start) {
    frontier[start] = 1;
}